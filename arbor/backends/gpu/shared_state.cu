#include "hip/hip_runtime.h"
// GPU kernels and wrappers for shared state methods.

#include <cstdint>

#include <backends/event.hpp>
#include <backends/event_stream_state.hpp>

#include <arbor/gpu/gpu_api.hpp>
#include <arbor/gpu/gpu_common.hpp>

namespace arb {
namespace gpu {

namespace kernel {

// Vector/scalar addition: x[i] += v ∀i
template <typename T>
__global__ void add_scalar(unsigned n,
                           T* __restrict__ const x,
                           arb_value_type v) {
    unsigned tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid < n) x[tid] += v;
}

__global__ void take_samples_impl(const raw_probe_info* __restrict__ const begin_marked,
                                  const raw_probe_info* __restrict__ const end_marked,
                                  const arb_value_type time,
                                  arb_value_type* __restrict__ const sample_time,
                                  arb_value_type* __restrict__ const sample_value) {
    const unsigned tid = threadIdx.x + blockIdx.x*blockDim.x;
    const auto ptr = begin_marked + tid;
    if (ptr < end_marked) {
        auto offset = ptr->offset;
        sample_time[offset] = time;
        sample_value[offset] = ptr->handle? *ptr->handle : 0;
    }
}

} // namespace kernel

void add_scalar(std::size_t n, arb_value_type* data, arb_value_type v) {
    launch_1d(n, 128, kernel::add_scalar<arb_value_type>, n, data, v);
}

void take_samples_impl(
    const event_stream_state<raw_probe_info>& s,
    const arb_value_type& time, arb_value_type* sample_time, arb_value_type* sample_value) {
    launch_1d(s.size(), 128, kernel::take_samples_impl, s.begin_marked, s.end_marked, time, sample_time, sample_value);
}

} // namespace gpu
} // namespace arb
