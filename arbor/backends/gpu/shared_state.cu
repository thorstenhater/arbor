#include "hip/hip_runtime.h"
// GPU kernels and wrappers for shared state methods.

#include <cstdint>

#include <backends/event.hpp>
#include <backends/multi_event_stream_state.hpp>

#include "gpu_api.hpp"
#include "gpu_common.hpp"

namespace arb {
namespace gpu {

namespace kernel {

template <typename T>
__global__ void update_time_to_impl(unsigned n,
                                    T* __restrict__ const time_to,
                                    const T* __restrict__ const time,
                                    T dt,
                                    T tmax) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        auto t = time[i]+dt;
        time_to[i] = t<tmax? t: tmax;
    }
}

template <typename T, typename I>
__global__ void add_gj_current_impl(unsigned n,
                                    const T* __restrict__ const gj_info,
                                    const I* __restrict__ const voltage,
                                    I* __restrict__ const current_density) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        auto gj = gj_info[i];
        auto curr = gj.weight * (voltage[gj.loc.second] - voltage[gj.loc.first]); // nA

        gpu_atomic_sub(current_density + gj.loc.first, curr);
    }
}

// Vector/scalar addition: x[i] += v ∀i
template <typename T>
__global__ void add_scalar(unsigned n,
                           T* __restrict__ const x,
                           fvm_value_type v) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        x[i] += v;
    }
}

template <typename T, typename I>
__global__ void set_dt_impl(      T* __restrict__ dt_intdom,
                            const T* __restrict__ time_to,
                            const T* __restrict__ time,
                            const unsigned ncomp,
                                  T* __restrict__ dt_comp,
                            const I* __restrict__ cv_to_intdom) {
    auto idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < ncomp) {
        const auto ind = cv_to_intdom[idx];
        const auto dt = time_to[ind] - time[ind];
        dt_intdom[ind] = dt;
        dt_comp[idx] = dt;
    }
}

__global__ void take_samples_impl(
    multi_event_stream_state<raw_probe_info> s,
    const fvm_value_type* __restrict__ const time,
    fvm_value_type* __restrict__ const sample_time,
    fvm_value_type* __restrict__ const sample_value)
{
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<s.n) {
        auto begin = s.ev_data+s.begin_offset[i];
        auto end = s.ev_data+s.end_offset[i];
        for (auto p = begin; p!=end; ++p) {
            sample_time[p->offset] = time[i];
            sample_value[p->offset] = *p->handle;
        }
    }
}

} // namespace kernel

using impl::block_count;

void add_scalar(std::size_t n, fvm_value_type* data, fvm_value_type v) {
    if (!n) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(n, block_dim);
    kernel::add_scalar<<<nblock, block_dim>>>(n, data, v);
}

void update_time_to_impl(
    std::size_t n, fvm_value_type* time_to, const fvm_value_type* time,
    fvm_value_type dt, fvm_value_type tmax)
{
    if (!n) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(n, block_dim);
    kernel::update_time_to_impl<<<nblock, block_dim>>>(n, time_to, time, dt, tmax);
}

void set_dt_impl(
    fvm_size_type nintdom, fvm_size_type ncomp, fvm_value_type* dt_intdom, fvm_value_type* dt_comp,
    const fvm_value_type* time_to, const fvm_value_type* time, const fvm_index_type* cv_to_intdom)
{
    if (!nintdom || !ncomp) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(ncomp, block_dim);
    kernel::set_dt_impl<<<nblock, block_dim>>>(dt_intdom, time_to, time, ncomp, dt_comp, cv_to_intdom);
}

void add_gj_current_impl(
    fvm_size_type n_gj, const fvm_gap_junction* gj_info, const fvm_value_type* voltage, fvm_value_type* current_density)
{
    if (!n_gj) return;

    constexpr int block_dim = 128;
    int nblock = block_count(n_gj, block_dim);
    kernel::add_gj_current_impl<<<nblock, block_dim>>>(n_gj, gj_info, voltage, current_density);
}

void take_samples_impl(
    const multi_event_stream_state<raw_probe_info>& s,
    const fvm_value_type* time, fvm_value_type* sample_time, fvm_value_type* sample_value)
{
    if (!s.n_streams()) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(s.n_streams(), block_dim);
    kernel::take_samples_impl<<<nblock, block_dim>>>(s, time, sample_time, sample_value);
}

} // namespace gpu
} // namespace arb
