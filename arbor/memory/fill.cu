#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>

#include <cstdint>
#include <iostream>

namespace arb {
namespace gpu {

template <typename T, typename I>
__global__
void fill_kernel(T* __restrict__ const v, T value, I n) {
    auto tid = threadIdx.x + blockDim.x*blockIdx.x;

    if(tid < n) {
        v[tid] = value;
    }
}

__global__
void fill_kernel_vec(std::uint64_t* __restrict__ const v, std::uint64_t value, std::size_t n) {
    auto tid = threadIdx.x + blockDim.x*blockIdx.x;
    auto v4 = reinterpret_cast<ulong4*>(v);
    ulong4 value4 = make_ulong4(value, value, value, value);
    size_t n4 = n/4;
    // Bulk write
    if(tid < n4) v4[tid] = value4;
    // First thread handles the remainder at the tail end
    if(tid == 0) {
        for (int ix = 0; ix < n%4; ++ix) {
            v[ix + 4*n4] = value;
        }
    }
}

constexpr static int block_size = 128;

void fill8(uint8_t* v, uint8_t value, std::size_t n) {
    launch_1d(n, block_size, fill_kernel<uint8_t, std::size_t>, v, value, n);
}

void fill16(uint16_t* v, uint16_t value, std::size_t n) {
    launch_1d(n, block_size, fill_kernel<uint16_t, std::size_t>, v, value, n);
}

void fill32(uint32_t* v, uint32_t value, std::size_t n) {
    launch_1d(n, block_size, fill_kernel<uint32_t, std::size_t>, v, value, n);
}

void fill64(uint64_t* v, uint64_t value, std::size_t n) {
    if (n >= 4) {
        auto len = (n + 3)/4;
        std::cout << "[VEC] val=" << value << " ptr=" << v << " cnt=" << n << '\n';
        launch_1d(len, block_size, fill_kernel_vec, v, value, n);
    }
    else {
        std::cout << "[SCL] val=" << value << " ptr=" << v << " cnt=" << n << '\n';
        launch_1d(n, block_size, fill_kernel_vec, v, value, n);
    }
}

} // namespace gpu
} // namespace arb
